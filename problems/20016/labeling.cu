#include "hip/hip_runtime.h"
#include "labeling.h"

#define BLOCK_SIZE 512

__global__
void labling_kernel(const char *cuStr, int *cuPos, const int strLen) {
    __shared__ int local_pos[BLOCK_SIZE];

    int pos_index = threadIdx.x + blockIdx.x*blockDim.x;
    int index = threadIdx.x;

    // thrust::tabulate, mark_spaces
    local_pos[index] = (cuStr[pos_index] > ' ') ? -1 : pos_index;
    __syncthreads();

    // thrust::inclusive_scan, thrust::maximum<int>
    for (int offset = 1; offset <= index; offset *= 2) {
        if (local_pos[index] < local_pos[index-offset]) {
            local_pos[index] = local_pos[index-offset];
        }
        __syncthreads();
    }
    __syncthreads();

    // thrust::tabulate, sub_offset
    if (pos_index < strLen) {
        cuPos[pos_index] = pos_index - local_pos[index];
    }
}

__global__
void patch_kernel(int *cuPos, const int strLen) {
    int pos_index = threadIdx.x + blockIdx.x*blockDim.x;
    int index = threadIdx.x;

    if (pos_index >= strLen) {
        return;
    }

    // cross blocks
    if (blockIdx.x > 0 && cuPos[pos_index] == (index+1)) {
        cuPos[pos_index] += cuPos[blockIdx.x*blockDim.x-1];
    }
}

void labeling(const char *cuStr, int *cuPos, int strLen) {
    int n_blocks = (strLen + BLOCK_SIZE-1) / BLOCK_SIZE;
    labling_kernel<<<n_blocks, BLOCK_SIZE>>>(cuStr, cuPos, strLen);
    patch_kernel<<<n_blocks, BLOCK_SIZE>>>(cuPos, strLen);
}
