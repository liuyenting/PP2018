#include "hip/hip_runtime.h"
#include "labeling.h"

__global__
void count_position_kernel(
    const char *input,
    int *output,
    const int n
) {
    for (
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        i < n;
        i += blockDim.x * gridDim.x
    ) {
        if ((input[i] != ' ') && ((i == 0) || (input[i-1] == ' '))) {
            int j = i, c = 1;
            do {
                output[j++] = c++;
            } while ((input[j] != ' ') && (j < n));
        }
    }
}

void labeling(const char *text, int *pos, int text_size) {
    int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);

    hipMemset(pos, 0, text_size*sizeof(int));
    count_position_kernel<<<32*numSMs, 256>>>(text, pos, text_size);
}
