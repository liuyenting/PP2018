#include "hip/hip_runtime.h"
#include "labeling.h"

#include <cstdio>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

struct is_alphabet {
    __device__
    int operator()(const char c) const {
        return (c != '\n') ? 1 : 0;
    }
};

void CountPosition1(const char *text, int *pos, int text_size)
{
    thrust::transform(
        thrust::device,
        text,
        text + text_size,
        pos,
        is_alphabet()
    );

    thrust::inclusive_scan_by_key(
        thrust::device,
        pos,
        pos + text_size,
        pos,
        pos
    );
}

__global__
void count_position_kernel(
    const char *input,
    int *output,
    const int n
) {
    for (
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        i < n;
        i += blockDim.x * gridDim.x
    ) {
        if ((input[i] != ' ') && ((i == 0) || (input[i-1] == ' '))) {
            int j = i, c = 1;
            do {
                output[j++] = c++;
            } while ((input[j] != ' ') && (j < n));
        }
    }
}

void labeling(const char *text, int *pos, int text_size) {
    int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);

    hipMemset(pos, 0, text_size*sizeof(int));
    count_position_kernel<<<32*numSMs, 256>>>(text, pos, text_size);
}
