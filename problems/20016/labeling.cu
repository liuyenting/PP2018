#include "hip/hip_runtime.h"
#include "labeling.h"

#include <cstdio>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

struct is_alphabet {
    __device__
    int operator()(const char c) const {
        return (c != '\n') ? 1 : 0;
    }
};

void CountPosition1(const char *text, int *pos, int text_size)
{
    thrust::transform(
        thrust::device,
        text,
        text + text_size,
        pos,
        is_alphabet()
    );

    thrust::inclusive_scan_by_key(
        thrust::device,
        pos,
        pos + text_size,
        pos,
        pos
    );
}

namespace lab2 {

__global__
void count_position_kernel(
    const char *input,
    int *output,
    const int n
) {
    for (
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        i < n;
        i += blockDim.x * gridDim.x
    ) {
        if ((input[i] != '\n') && ((i == 0) || (input[i-1] == '\n'))) {
            int j = i, c = 1;
            do {
                output[j++] = c++;
            } while ((input[j] != '\n') && (j < n));
        }
    }
}

}

void labeling(const char *text, int *pos, int text_size) {
    int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);

    hipMemset(pos, 0, text_size*sizeof(int));
    lab2::count_position_kernel<<<32*numSMs, 256>>>(text, pos, text_size);
}
